#include "hip/hip_runtime.h"
#include "Regression.h"
#include <ctime>

int getNumberOfWeights (NeuralNet& neuralNet);
void updateNetworkWeights (NeuralNet& neuralNet);
void calcGrad (NeuralNet& neuralNet);
__global__ void gradFromOutputLayer (double* d_dydw, double* singleGradient, unsigned int* direction, const unsigned int size, const unsigned int numOutputs, const unsigned int batchSize, const unsigned int weightsSize);
__global__ void gradFromSecondHidden (double* d_dydx, double* d_dydw, double* singleGradient, unsigned int* direction, const unsigned int neuronsPerLayer, const unsigned int numOutputs, const unsigned int batchSize, const unsigned int weightsSize);
__global__ void gradFromFirstHidden (double* d_dydxOut, double* d_dydxHid, double* d_dydw, double* singleGradient, unsigned int* direction, const unsigned int iSize, const unsigned int jSize, const unsigned int numOutputs, const unsigned int batchSize, const unsigned int weightsSize);

using namespace std;

Datum::Datum (double lDist, double fDist, double rDist) :
  fDist (fDist), lDist (lDist), rDist (rDist)
{}

bool running = true;
static double scale = 1 / 10000;
Datum* dataVector = new Datum[dataSize];
Direction* resultsVector = new Direction[dataSize];
unsigned int weightsSize = 0;
static double* d_weights = nullptr;
static double* d_gradient = nullptr;
static double* d_input = nullptr;
static unsigned int* d_direction = nullptr;

__global__ void incrementWeights (double* d_weights, double* d_gradient)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  d_weights[i] += d_gradient[i];
}

void gradientDescentLoop (NeuralNet& neuralNet, int batches)
{
  weightsSize = getNumberOfWeights (neuralNet);
  CUDA_CALL (hipMalloc (&d_weights, weightsSize * sizeof (double)));
  CUDA_CALL (hipMalloc (&d_gradient, weightsSize * sizeof (double)));
  CUDA_CALL (hipMalloc (&d_input, dataSize * 4 * sizeof (double)));
  CUDA_CALL (hipMalloc (&d_direction, dataSize * sizeof (unsigned int)));

  CUDA_CALL (hipMemcpy (d_input, dataVector, dataSize * 4 * sizeof (double), hipMemcpyHostToDevice));
  CUDA_CALL (hipMemcpy (d_direction, resultsVector, dataSize * sizeof (unsigned int), hipMemcpyHostToDevice));
  getNetworkWeights (neuralNet);
  while (running)
  {
    cout << "Continuing, not converged yet..." << endl;
    for (int batch = 1; batch <= batches; batch++)
    {
      std::clock_t start;
      double duration;

      start = std::clock ();

      calcGrad (neuralNet);
      incrementWeights << <(weightsSize + 31) / 32, 32 >> > (d_weights, d_gradient);
      CUDA_GET_ERROR ();
      updateNetworkWeights (neuralNet);
      hipDeviceSynchronize();

      duration = (std::clock () - start) / (double)CLOCKS_PER_SEC;
      cout << "This batch took " << duration << endl;
    }
    cout << "Saving Weights..." << endl;
    neuralNet.saveWeights ("weights.dat");
  }
  cout << "Regression has stopped." << endl;
  hipFree (d_weights);
  hipFree (d_gradient);
  hipFree (d_input);
  hipFree (d_direction);
}

void generateData ()
{
  int counter = 0;
  for (int lDist = 1; lDist <= 150; lDist += 4)
  {
    for (int rDist = 1; rDist <= 150; rDist += 4)
    {
      for (int fDist = 1; fDist <= 150; fDist += 4)
      {
        if (fDist > 100)
        {
          dataVector[counter] = Datum (lDist, fDist, rDist);
          resultsVector[counter] = Direction::forward;
          ++counter;
        }
        if (fDist < 50 && rDist > lDist)
        {
          dataVector[counter] = Datum (lDist, fDist, rDist);
          resultsVector[counter] = Direction::right;
          ++counter;
        }
        if (fDist < 50 && lDist > rDist)
        {
          dataVector[counter] = Datum (lDist, fDist, rDist);
          resultsVector[counter] = Direction::right;
          ++counter;
        }
      }
    }
  }
}

void cleanData ()
{
  delete[] dataVector;
  delete[] resultsVector;
}

int getNumberOfWeights (NeuralNet& neuralNet)
{
  int numberOfWeights = 0;
  for_each (begin (neuralNet.layers), end (neuralNet.layers),
    [&](HiddenLayer& layer) {
    numberOfWeights += layer.numberOfNeurons * (layer.numberOfInputs + 1);
  });
  OutputLayer& outputLayer = neuralNet.outputLayer;
  numberOfWeights += outputLayer.numberOfNeurons * (outputLayer.numberOfInputs + 1);
  return numberOfWeights;
}

void getNetworkWeights (NeuralNet& neuralNet)
{
  CUDA_CALL (hipMemset (d_weights, 0, weightsSize * sizeof (double)));
  double* d_weightsCopy = d_weights;
  for_each (begin (neuralNet.layers), end (neuralNet.layers),
    [&](HiddenLayer& layer) {
    hipMemcpy (d_weightsCopy, layer.d_weightMatrix, layer.numberOfNeurons * (layer.numberOfInputs + 1) * sizeof (double), hipMemcpyDeviceToDevice);
    d_weightsCopy += layer.numberOfNeurons * (layer.numberOfInputs + 1);
  });
  hipMemcpy (d_weightsCopy, neuralNet.outputLayer.d_weightMatrix, neuralNet.outputLayer.numberOfNeurons * (neuralNet.outputLayer.numberOfInputs + 1) * sizeof (double), hipMemcpyDeviceToDevice);
}

void updateNetworkWeights (NeuralNet& neuralNet)
{
  double* d_weightsPtr = d_weights;
  for_each (begin (neuralNet.layers), end (neuralNet.layers),
    [&d_weightsPtr](HiddenLayer& layer) {
    CUDA_CALL (hipMemcpy (layer.d_weightMatrix, d_weightsPtr, layer.numberOfNeurons * (layer.numberOfInputs + 1) * sizeof (double), hipMemcpyDeviceToDevice));
    d_weightsPtr += layer.numberOfNeurons * (layer.numberOfInputs + 1);
  });
  CUDA_CALL (hipMemcpy (neuralNet.outputLayer.d_weightMatrix, d_weightsPtr, neuralNet.outputLayer.numberOfNeurons * (neuralNet.outputLayer.numberOfInputs + 1) * sizeof (double), hipMemcpyDeviceToDevice));
}

//Only works for exactly 2 hidden layers.
void calcGrad (NeuralNet& neuralNet)
{
  neuralNet.processInput ((double*)d_input);

  const unsigned int iSize = neuralNet.neuronsPerLayer;
  const unsigned int jSize = neuralNet.numberOfInputs + 1;
  const unsigned int numOutputs = neuralNet.numberOfOutputs;
  const unsigned int yDim = 32;

  gradFromFirstHidden <<<{jSize, (dataSize + yDim - 1) / yDim}, { iSize, yDim }, yDim*iSize*sizeof(double)>>> (neuralNet.outputLayer.d_dydx, neuralNet.layers[1].d_dydx, neuralNet.layers[0].d_dydw,
    d_gradient, d_direction, iSize, jSize, numOutputs, dataSize, weightsSize);
  CUDA_GET_ERROR ();

  double* layerTwoGrad = d_gradient + neuralNet.neuronsPerLayer * (neuralNet.numberOfInputs + 1);
  gradFromSecondHidden <<< {jSize, (dataSize + yDim - 1) / yDim}, { iSize, yDim }, iSize*yDim*sizeof(double) >>> (neuralNet.outputLayer.d_dydx, neuralNet.layers[1].d_dydw, 
    layerTwoGrad, d_direction, neuralNet.neuronsPerLayer, numOutputs, dataSize, weightsSize);
  CUDA_GET_ERROR ();

  double* outGrad = layerTwoGrad + neuralNet.neuronsPerLayer * (neuralNet.neuronsPerLayer + 1);
  unsigned int size = numOutputs * (neuralNet.neuronsPerLayer + 1);
  gradFromOutputLayer <<< {(size + 31) / 32, (dataSize + 7) / 8}, { 32, 8 } >>> (neuralNet.outputLayer.d_dydw, outGrad, d_direction, size, numOutputs, dataSize, weightsSize);
  CUDA_GET_ERROR ();
}

__global__ void gradFromOutputLayer (double* d_dydw, double* grad, unsigned int* direction, const unsigned int size, const unsigned int numOutputs, const unsigned int batchSize, const unsigned int weightsSize)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int batch = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < size && batch < batchSize)
  {
    atomicAdd(grad + i, d_dydw[batch * numOutputs * size + direction[batch] * size + i]);
  }
}

__global__ void gradFromSecondHidden (double* d_dydx, double* d_dydw, double* grad, unsigned int* direction, const unsigned int neuronsPerLayer,
  const unsigned int numOfOutputs, const unsigned int batchSize, const unsigned int weightsSize)
{
  extern __shared__ double s[];
  const int iSize = neuronsPerLayer; //blockDim.x
  const int jSize = neuronsPerLayer + 1;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int batch = blockIdx.y * blockDim.y + threadIdx.y;
  double gradVal = 0;
  s[threadIdx.y*iSize + threadIdx.x] = d_dydx[batch*numOfOutputs*iSize + direction[batch]*iSize + threadIdx.x];
  __syncthreads();
  if (j < jSize * iSize && batch < batchSize)
  {
    #pragma unroll
    for (int k = 0; k < iSize; ++k)
    {
      gradVal += s[threadIdx.y*iSize + k] //d_dydx[batch * numOfOutputs * iSize + direction[batch] * iSize + k]
      * d_dydw[batch * iSize * iSize * jSize + k * iSize * jSize + j];
    }
    atomicAdd(grad+j, gradVal);
  }
}

__global__ void gradFromFirstHidden (double* d_dydxOut, double* d_dydxHid, double* d_dydw, double* grad, unsigned int* direction, const unsigned int iSize, const unsigned int jSize,
  const unsigned int numOutputs, const unsigned int batchSize, const unsigned int weightsSize)
{
  const int j = blockIdx.x * blockDim.y + threadIdx.x;
  const int batch = blockIdx.y * blockDim.y + threadIdx.y;
  extern __shared__ double outyx[];

  if(threadIdx.x < iSize && batch < batchSize)
  {
    #pragma unroll    
    for (int k = 0; k < iSize; k++)
    {
      outyx[threadIdx.y*iSize + threadIdx.x] += d_dydxHid[batch*iSize*jSize + k*iSize + threadIdx.x] *  d_dydxOut[batch * numOutputs * iSize + direction[batch] * iSize + k];
    }
  }

  __syncthreads();
  double gradVal = 0;
  if (j < iSize * jSize && batch < batchSize)
  {
    #pragma unroll
    for(int l = 0; l < iSize; ++l)
    {
      gradVal += outyx[threadIdx.y*iSize + l] * d_dydw[batch * iSize * jSize * iSize + l * iSize * jSize + j];
    }
    atomicAdd(grad + j, gradVal / 1000);
  }  

    // for (int k = 0; k < iSize; k++)
    // {
    //   for (int l = 0; l < iSize; l++)
    //   {
    //     singleGradient[batch * weightsSize + j] +=
    //       d_dydxOut[batch * numOutputs * iSize + direction[batch] * iSize + k] * d_dydxHid[batch * iSize * iSize + k * iSize + l] * d_dydw[batch * iSize * jSize * iSize + l * iSize * jSize + j];
    //   }
    // }       
}
